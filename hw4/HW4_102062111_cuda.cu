#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <iostream>
#include <string.h>

#define INF 1e9
#define H2D hipMemcpyHostToDevice
#define D2H hipMemcpyDeviceToHost
#define timer(type) \
    hipEventCreate(&type##_start); \
    hipEventCreate(&type##_stop);

#define record(type) \
    hipEventRecord(type)

#define elapsed(type, start, stop) \
    hipEventElapsedTime(&type, start, stop)

#define sync(type) \
    hipEventSynchronize(type)

#define MASK_N 2
#define MASK_X 5
#define MASK_Y 5
#define SCALE  8

#define cudaCheckErrors(msg) \
    do { \
        hipError_t __err = hipGetLastError(); \
        if (__err != hipSuccess) { \
            fprintf(stderr, "Fatal error: %s (%s at %s:%d)\n", \
                msg, hipGetErrorString(__err), \
                __FILE__, __LINE__); \
            fprintf(stderr, "*** FAILED - ABORTING\n"); \
            exit(1); \
        } \
    } while (0)

using namespace std;

void DisplayHeader()
{
    const int kb = 1024;
    const int mb = kb * kb;
    wcout << "NBody.GPU" << endl << "=========" << endl << endl;

    //wcout << "CUDA version:   v" << CUDART_VERSION << endl;    
    //wcout << "Thrust version: v" << THRUST_MAJOR_VERSION << "." << THRUST_MINOR_VERSION << endl << endl; 

    int devCount;
    hipGetDeviceCount(&devCount);
    wcout << "CUDA Devices: " << endl << endl;

    for(int i = 0; i < devCount; ++i)
    {
        hipDeviceProp_t props;
        hipGetDeviceProperties(&props, i);
        wcout << i << ": " << props.name << ": " << props.major << "." << props.minor << endl;
        wcout << "  Global memory:   " << props.totalGlobalMem / mb << "mb" << endl;
        wcout << "  Shared memory:   " << props.sharedMemPerBlock / kb << "kb" << endl;
        wcout << "  Constant memory: " << props.totalConstMem / kb << "kb" << endl;
        wcout << "  Block registers: " << props.regsPerBlock << endl << endl;

        wcout << "  Warp size:         " << props.warpSize << endl;
        wcout << "  Threads per block: " << props.maxThreadsPerBlock << endl;
        wcout << "  Max block dimensions: [ " << props.maxThreadsDim[0] << ", " << props.maxThreadsDim[1]  << ", " << props.maxThreadsDim[2] << " ]" << endl;
        wcout << "  Max grid dimensions:  [ " << props.maxGridSize[0] << ", " << props.maxGridSize[1]  << ", " << props.maxGridSize[2] << " ]" << endl;
        wcout << endl;
    }
}


__global__ void floyd_warshall(int *d, int blockSize, int length, 
                            int XIndex, int YIndex, int kk) {
    int ii = blockSize * XIndex + blockIdx.x * blockDim.x + threadIdx.x;
    int jj = blockSize * YIndex + blockIdx.y * blockDim.y + threadIdx.y; 

    int dij = d[ii * length + jj];
    int dik = d[ii * length + kk];
    int dkj = d[kk * length + jj];
    
    if (dij > dik + dkj)
       d[ii * length + jj] = dik + dkj;   
}


int main(int argc, char **argv) {
    hipEvent_t total_start, total_stop;
    hipEvent_t com_start, com_stop;
    hipEvent_t mem_start, mem_stop;
    hipEvent_t io_start, io_stop;
    
    timer(total);
    timer(com);
    timer(mem);
    timer(io); 
    
    hipEventRecord(total_start); 
        
    float total, compute, memory, IO;
    float mem_part, IO_part;
    total = compute = memory = IO = 0; 
        
    FILE *fin = fopen(argv[1], "r");
    FILE *fout = fopen(argv[2], "w");
    int blockSize = atoi(argv[3]);
     
    int N, M;
    int *edge;
    int *cuda_edge;
    
    fscanf(fin, "%d %d", &N, &M);
    int gridSize = N % blockSize ? N / blockSize + 1 : N / blockSize;
    int length = blockSize * gridSize + 1;
    edge = new int[length * length];
    
    //fprintf(stderr, "grid = %d, block = %d\n", gridSize, blockSize);

    for (int i = 0; i < length; ++i) {
        for (int j = 0; j < length; ++j)
            edge[i * length + j] = INF;
        edge[i * length + i] = 0;
    }


    record(io_start);
    while (M--) {
        int a, b, w;
        fscanf(fin, "%d %d %d", &a, &b, &w);
        a = a - 1;
        b = b - 1;
        edge[a * length + b] = w;
    }
    record(io_stop);
    sync(io_stop);
    elapsed(IO_part, io_start, io_stop);
    IO += IO_part;

    hipSetDevice(0);
    
    hipMalloc((void**)&cuda_edge, sizeof(int) * length * length);
    cudaCheckErrors("malloc cuda_edge");

    record(mem_start);     
    hipMemcpy(cuda_edge, edge, sizeof(int) * length * length, H2D);
    cudaCheckErrors("copy cuda_edge");
    record(mem_stop);
    sync(mem_stop);
    elapsed(mem_part, mem_start, mem_stop);
    memory += mem_part;

    // Now only hangle N = 3200 testcase
    // size_t sharedSize = 8 * 8;
    int blockNum = (N + blockSize - 1) / blockSize;
    int blockDimension = 8; 
    gridSize = blockSize / blockDimension;
    int gridFactor = 1024 / blockSize ;
    gridFactor *= gridFactor;


    dim3 blocks(gridSize, gridSize);
    dim3 threads(blockDimension, blockDimension);
    
    dim3 blockCol(gridSize * gridFactor, gridSize);
    dim3 blockRow(gridSize, gridSize * gridFactor);
    int remainBegin = (blockNum / gridFactor) * gridFactor ;  
    int remain = blockNum - remainBegin;
    dim3 blockColRemain(gridSize * remain, gridSize);
    dim3 blockRowRemain(gridSize, gridSize * remain);
    //wcout << "blocknum = " << blockNum << endl; 
    
    hipEventRecord(com_start);
    for (int k = 0; k < blockNum; ++k) {
        //wcout << "k = " << k << endl;
        // phase one
        {
            for (int cur = 0; cur < blockSize; ++cur) {
                //wcout << "(" << cur << "/" << blockSize << endl;
                floyd_warshall<<<blocks, threads>>>
                    (cuda_edge, blockSize, length, k, k, k * blockSize + cur);
            
                cudaCheckErrors("phase one");
            }
        }
        // phase two
        {
            // Column 
            for (int i = 0; i < blockNum - remain; i = i + gridFactor) {
                for (int cur = 0; cur < blockSize; cur++) {
                    floyd_warshall<<<blockCol, threads>>>
                            (cuda_edge, blockSize, length, i, k, k * blockSize + cur);
                    
                    cudaCheckErrors("phase two column main");
                }       
            }
        
            if (remainBegin < blockNum)
                for (int cur = 0; cur < blockSize; cur++) {
                    floyd_warshall<<<blockColRemain, threads>>>
                            (cuda_edge, blockSize, length, remainBegin, k, k * blockSize + cur);
                    cudaCheckErrors("phase two column remain");
                }
            // Row 
            for (int j = 0; j < blockNum - remain; j = j + gridFactor) {
                for (int cur = 0; cur < blockSize; ++cur) {
                    floyd_warshall<<<blockRow, threads>>>
                            (cuda_edge, blockSize, length, k, j, k * blockSize + cur);
                    cudaCheckErrors("phase two row main");
                }
            }
            if (remainBegin < blockNum)
                for (int cur = 0; cur < blockSize; cur++) {
                    floyd_warshall<<<blockRowRemain, threads>>>
                            (cuda_edge, blockSize, length, k, remainBegin, k * blockSize + cur);
                    cudaCheckErrors("phase two row remain");
                }
        }   
        
        //phase three
        {
            
            for (int i = 0; i < blockNum; i++) {
                for (int j = 0; j < blockNum - remain; j = j + gridFactor) {
                    for (int cur = 0; cur < blockSize; ++cur) {
                        floyd_warshall<<<blockRow, threads>>>
                                (cuda_edge, blockSize, length, i, j, k * blockSize + cur);
                        cudaCheckErrors("phase three row main");
                    }
                }
                if (remainBegin < blockNum)
                    for (int cur = 0; cur < blockSize; cur++) {
                        floyd_warshall<<<blockRowRemain, threads>>>
                                (cuda_edge, blockSize, length, i, remainBegin, k * blockSize + cur);
                        cudaCheckErrors("phase three row remain");
                    }
            }
            
            /* 
            for (int i = 0; i < blockNum; ++i) {
                for (int j = 0; j < blockNum; ++j)
                    if (i != k && j != k)
                        for (int cur = 0; cur < blockSize; ++cur)
                            floyd_warshall<<<blocks, threads>>>
                                    (cuda_edge, blockSize, length, i, j, k * blockSize + cur);
            }
            */
       
        }
        
    }
    hipDeviceSynchronize();
    hipEventRecord(com_stop);
    hipEventSynchronize(com_stop);
    hipEventElapsedTime(&compute, com_start, com_stop);

     
    record(mem_start);
    hipMemcpy(edge, cuda_edge, sizeof(int) * length * length, D2H);
    record(mem_stop);
    sync(mem_stop);
    elapsed(mem_part, mem_start, mem_stop);
    memory += mem_part; 
    

    record(io_start);
    for (int i = 0; i < N; ++i) {
        for (int j = 0; j < N - 1; ++j) {
            if (edge[i * length + j] == INF)
                fprintf(fout, "INF ");
            else
                fprintf(fout, "%d ", edge[i * length + j]);
        }
        if (edge[i * length + N - 1] == INF)
            fprintf(fout, "INF\n");
        else
            fprintf(fout, "%d\n", edge[i * length + N - 1]);
    }
    record(io_stop);
    sync(io_stop);
    elapsed(IO_part, io_start, io_stop);
    IO += IO_part;

    hipEventRecord(total_stop);
    hipEventSynchronize(total_stop);
    hipEventElapsedTime(&total, total_start, total_stop);
    
    fprintf(stderr, "\n\n");
    fprintf(stderr, "TOTAL = %f\n", total);
    fprintf(stderr, "COMPUTE = %f\n", compute);
    fprintf(stderr, "MEMORY = %f\n", memory);
    fprintf(stderr, "IO = %f\n", IO);
    
    return 0;
}
