#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <iostream>
#include <string.h>

#define INF 1e9
#define H2D hipMemcpyHostToDevice
#define D2H hipMemcpyDeviceToHost
#define MASK_N 2
#define MASK_X 5
#define MASK_Y 5
#define SCALE  8

#define cudaCheckErrors(msg) \
    do { \
        hipError_t __err = hipGetLastError(); \
        if (__err != hipSuccess) { \
            fprintf(stderr, "Fatal error: %s (%s at %s:%d)\n", \
                msg, hipGetErrorString(__err), \
                __FILE__, __LINE__); \
            fprintf(stderr, "*** FAILED - ABORTING\n"); \
            exit(1); \
        } \
    } while (0)

using namespace std;

void DisplayHeader()
{
    const int kb = 1024;
    const int mb = kb * kb;
    wcout << "NBody.GPU" << endl << "=========" << endl << endl;

    //wcout << "CUDA version:   v" << CUDART_VERSION << endl;    
    //wcout << "Thrust version: v" << THRUST_MAJOR_VERSION << "." << THRUST_MINOR_VERSION << endl << endl; 

    int devCount;
    hipGetDeviceCount(&devCount);
    wcout << "CUDA Devices: " << endl << endl;

    for(int i = 0; i < devCount; ++i)
    {
        hipDeviceProp_t props;
        hipGetDeviceProperties(&props, i);
        wcout << i << ": " << props.name << ": " << props.major << "." << props.minor << endl;
        wcout << "  Global memory:   " << props.totalGlobalMem / mb << "mb" << endl;
        wcout << "  Shared memory:   " << props.sharedMemPerBlock / kb << "kb" << endl;
        wcout << "  Constant memory: " << props.totalConstMem / kb << "kb" << endl;
        wcout << "  Block registers: " << props.regsPerBlock << endl << endl;

        wcout << "  Warp size:         " << props.warpSize << endl;
        wcout << "  Threads per block: " << props.maxThreadsPerBlock << endl;
        wcout << "  Max block dimensions: [ " << props.maxThreadsDim[0] << ", " << props.maxThreadsDim[1]  << ", " << props.maxThreadsDim[2] << " ]" << endl;
        wcout << "  Max grid dimensions:  [ " << props.maxGridSize[0] << ", " << props.maxGridSize[1]  << ", " << props.maxGridSize[2] << " ]" << endl;
        wcout << endl;
    }
}

__global__ void phaseOne(int *_d, int blockSize, int length, 
                                            int XIndex, int YIndex, int ZIndex) {
    int ii = blockSize * XIndex + threadIdx.x ;
    int jj = blockSize * YIndex + threadIdx.y ;

    __shared__ int d[40][40];

    int i = threadIdx.x;
    int j = threadIdx.y;
    
    d[threadIdx.x][threadIdx.y] = _d[ii * length + jj];
    
    __syncthreads();
    for (int k = 0; k < blockSize; ++k) {
        if (d[i][j] > d[i][k] + d[k][j])
            d[i][j] = d[i][k] + d[k][j];
        __syncthreads();
    }
    __syncthreads();
    
    _d[ii * length + jj] = d[i][j]; 
}

__global__ void phaseTwoRow(int *_d, int blockSize, int length, 
                                            int XIndex, int YIndex, int ZIndex) {
    int ii = blockSize * XIndex + threadIdx.x ;
    int jj = blockSize * YIndex + threadIdx.y ;
/*
    __shared__ int d[3][40][40];

    int i = threadIdx.x;
    int j = threadIdx.y;

    __shared__ int d[40][40];
    
    d[threadIdx.x][threadIdx.y] = _d[ii * length + jj];
    __syncthreads();
    
    for (int k = 0; k < blockSize; ++k) {
        if (d[i][j] > d[i][k] + d[k][j])
            d[i][j] = d[i][k] + d[k][j];
        __syncthreads();
    }
    __syncthreads();
    _d[ii * length + jj] = d[i][j]; 
*/
/*
    d[0][i][j] = _d[ii * length + jj];
    for (int k = 0, kk = ZIndex * blockSize; k < blockSize; ++k, ++kk) {
        if (j == 0)
            d[1][i][k] = _d[ii * length + kk];
        if (i == 0)
            d[2][k][j] = _d[kk * length + jj];
    }

    __syncthreads();
    for (int k = 0; k < blockSize; ++k) {
        if (d[0][i][j] > d[1][i][k] + d[2][k][j])
            d[0][i][j] = d[1][i][k] + d[2][k][j];
        __syncthreads();
    }
    __syncthreads();
    _d[ii * length + jj] = d[0][i][j];
*/
}
int main(int argc, char **argv) {
    
    FILE *fin = fopen(argv[1], "r");
    FILE *fout = fopen(argv[2], "w");
    int blockSize = atoi(argv[3]);
     
    int N, M;
    int *edge;
    int *cuda_edge;
    int *cuda_length;
    int *nodeNumber;
    int *index;
    
    fscanf(fin, "%d %d", &N, &M);
    int gridSize = N % blockSize ? N / blockSize + 1 : N / blockSize;
    int length = blockSize * gridSize + 1;
    edge = new int[length * length];
    
    fprintf(stderr, "grid = %d, block = %d\n", gridSize, blockSize);

    for (int i = 0; i < length; ++i) {
        for (int j = 0; j < length; ++j)
            edge[i * length + j] = INF;
        edge[i * length + i] = 0;
    }
    while (M--) {
        int a, b, w;
        fscanf(fin, "%d %d %d", &a, &b, &w);
        a = a - 1;
        b = b - 1;
        edge[a * length + b] = w;
    }
    hipSetDevice(0);
           
    hipMalloc((void**)&cuda_edge, sizeof(int) * length * length);
    cudaCheckErrors("malloc cuda_edge");

    hipMalloc((void**)&nodeNumber, sizeof(int));
    cudaCheckErrors("malloc cuda nodeNumber");

    hipMalloc((void**)&cuda_length, sizeof(int));
    cudaCheckErrors("malloc cuda_length");
    
    hipMalloc((void**)&index, sizeof(int));

    hipMemcpy(cuda_edge, edge, sizeof(int) * length * length, H2D);
    cudaCheckErrors("copy cuda_edge");
    
    hipMemcpy(nodeNumber, &N, sizeof(int), H2D);
    cudaCheckErrors("copy nodeNumber");
    
    hipMemcpy(cuda_length, &length, sizeof(int), H2D);
    cudaCheckErrors("copy cuda_length");

    // Now only hangle N = 3200 testcase
    size_t sharedSize = 3 * 40 * 40 ;
    gridSize = 1;
    dim3 blocks(gridSize, gridSize);
    dim3 threads(blockSize, blockSize);
    int blockNum = N / blockSize;
    wcout << "blocknum = " << blockNum << endl; 
    for (int k = 0; k < blockNum; ++k) {
        wcout << "k = " << k << endl;
        // phase one
        phaseOne<<<blocks, threads, sharedSize>>>
            (cuda_edge, blockSize, length, k, k, k);
 
        // phase two
    /*
        for (int i = 0; i < blockNum; ++i) {
            if (i != k) {
                phaseOne<<<blocks, threads, sharedSize>>>
                        (cuda_edge, blockSize, length, i, k, k);
            }
        }
     
        for (int j = 0; j < blockNum; ++j) {
            if (j != k) {
                floyd_warshall<<<blocks, threads, sharedSize>>>
                        (cuda_edge, blockSize, length, k, j, k);
            }
        }

        //phase three
        for (int i = 0; i < blockNum; ++i)
            for (int j = 0; j < blockNum; ++j)
                if (i != k && j != k)
                    floyd_warshall<<<blocks, threads, sharedSize>>>
                            (cuda_edge, blockSize, length, i, j, k);
    */
    /*
        fprintf(stderr, "k = %d\n", k);
        floyd_warshall<<<blocks, threads, sharedSize>>>
                    (cuda_edge ,blockSize, length, index);
    */
    }
    hipMemcpy(edge, cuda_edge, sizeof(int) * length * length, D2H);
    
    for (int i = 0; i < N; ++i) {
        for (int j = 0; j < N - 1; ++j) {
            if (edge[i * length + j] == INF)
                fprintf(fout, "INF ");
            else
                fprintf(fout, "%d ", edge[i * length + j]);
        }
        if (edge[i * length + N - 1] == INF)
            fprintf(fout, "INF\n");
        else
            fprintf(fout, "%d\n", edge[i * length + N - 1]);
    }
    return 0;
}
