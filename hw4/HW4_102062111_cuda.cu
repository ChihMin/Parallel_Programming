#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <iostream>
#include <string.h>

#define INF 1e9
#define H2D hipMemcpyHostToDevice
#define D2H hipMemcpyDeviceToHost
#define MASK_N 2
#define MASK_X 5
#define MASK_Y 5
#define SCALE  8

#define cudaCheckErrors(msg) \
    do { \
        hipError_t __err = hipGetLastError(); \
        if (__err != hipSuccess) { \
            fprintf(stderr, "Fatal error: %s (%s at %s:%d)\n", \
                msg, hipGetErrorString(__err), \
                __FILE__, __LINE__); \
            fprintf(stderr, "*** FAILED - ABORTING\n"); \
            exit(1); \
        } \
    } while (0)

using namespace std;

void DisplayHeader()
{
    const int kb = 1024;
    const int mb = kb * kb;
    wcout << "NBody.GPU" << endl << "=========" << endl << endl;

    //wcout << "CUDA version:   v" << CUDART_VERSION << endl;    
    //wcout << "Thrust version: v" << THRUST_MAJOR_VERSION << "." << THRUST_MINOR_VERSION << endl << endl; 

    int devCount;
    hipGetDeviceCount(&devCount);
    wcout << "CUDA Devices: " << endl << endl;

    for(int i = 0; i < devCount; ++i)
    {
        hipDeviceProp_t props;
        hipGetDeviceProperties(&props, i);
        wcout << i << ": " << props.name << ": " << props.major << "." << props.minor << endl;
        wcout << "  Global memory:   " << props.totalGlobalMem / mb << "mb" << endl;
        wcout << "  Shared memory:   " << props.sharedMemPerBlock / kb << "kb" << endl;
        wcout << "  Constant memory: " << props.totalConstMem / kb << "kb" << endl;
        wcout << "  Block registers: " << props.regsPerBlock << endl << endl;

        wcout << "  Warp size:         " << props.warpSize << endl;
        wcout << "  Threads per block: " << props.maxThreadsPerBlock << endl;
        wcout << "  Max block dimensions: [ " << props.maxThreadsDim[0] << ", " << props.maxThreadsDim[1]  << ", " << props.maxThreadsDim[2] << " ]" << endl;
        wcout << "  Max grid dimensions:  [ " << props.maxGridSize[0] << ", " << props.maxGridSize[1]  << ", " << props.maxGridSize[2] << " ]" << endl;
        wcout << endl;
    }
}

__global__ void phaseOne(int *_d, int blockSize, int length, 
                                            int XIndex, int YIndex, int ZIndex) {
    int ii = blockSize * XIndex + threadIdx.x ;
    int jj = blockSize * YIndex + threadIdx.y ;

    __shared__ int d[40][40];

    int i = threadIdx.x;
    int j = threadIdx.y;
    
    d[threadIdx.x][threadIdx.y] = _d[ii * length + jj];
    
    __syncthreads();
    for (int k = 0; k < blockSize; ++k) {
        if (d[i][j] > d[i][k] + d[k][j])
            d[i][j] = d[i][k] + d[k][j];
        __syncthreads();
    }
    __syncthreads();
    
    _d[ii * length + jj] = d[i][j]; 
}

__global__ void phaseTwoRow(int *_d, int blockSize, int length, 
                            int XIndex, int YIndex, int ZIndex) {
    int ii = blockSize * XIndex + threadIdx.x ;
    int jj = blockSize * YIndex + threadIdx.y ;
    
    int iii = blockSize * ZIndex + threadIdx.x;
    int jjj = blockSize * ZIndex + threadIdx.y; 
    
    __shared__ int d[2][40][40];

    int i = threadIdx.x;
    int j = threadIdx.y;

    d[0][i][j] = _d[ii * length + jj];
    d[1][i][j] = _d[iii * length + jjj]; 
    
    __syncthreads();
    
    for (int k = 0; k < blockSize; ++k) {
        if (d[0][i][j] > d[1][i][k] + d[0][k][j])
            d[0][i][j] = d[1][i][k] + d[0][k][j];
        __syncthreads();
    }
    __syncthreads();
    _d[ii * length + jj] = d[0][i][j]; 
}

__global__ void phaseTwoCol(int *_d, int blockSize, int length, 
                            int XIndex, int YIndex, int ZIndex) {
    int ii = blockSize * XIndex + threadIdx.x ;
    int jj = blockSize * YIndex + threadIdx.y ;
    
    int iii = blockSize * ZIndex + threadIdx.x;
    int jjj = blockSize * ZIndex + threadIdx.y; 
    
    __shared__ int d[2][40][40];

    int i = threadIdx.x;
    int j = threadIdx.y;

    d[0][i][j] = _d[ii * length + jj];
    d[1][i][j] = _d[iii * length + jjj]; 
    
    
    __syncthreads();
    
    for (int k = 0; k < blockSize; ++k) {
        if (d[0][i][j] > d[0][i][k] + d[1][k][j])
            d[0][i][j] = d[0][i][k] + d[1][k][j];
        __syncthreads();
    }
    __syncthreads();
    _d[ii * length + jj] = d[0][i][j]; 
}

__global__ void phaseThree(int *_d, int blockSize, int length, 
                            int XIndex, int YIndex, int ZIndex) {
    int ii = blockSize * XIndex + threadIdx.x ;
    int jj = blockSize * YIndex + threadIdx.y ;
    
    int iRow = blockSize * ZIndex + threadIdx.x;
    int jRow = blockSize * YIndex + threadIdx.y; 
    
    int iCol = blockSize * XIndex + threadIdx.x;
    int jCol = blockSize * ZIndex + threadIdx.y; 
    

    __shared__ int d[3][40][40];

    int i = threadIdx.x;
    int j = threadIdx.y;

    d[0][i][j] = _d[ii * length + jj];
    d[1][i][j] = _d[iRow * length + jRow]; 
    d[2][i][j] = _d[iCol * length + jCol];
    
    __syncthreads();
    
    for (int k = 0; k < blockSize; ++k) {
        if (d[0][i][j] > d[2][i][k] + d[1][k][j])
            d[0][i][j] = d[2][i][k] + d[1][k][j];
        __syncthreads();
    }
    __syncthreads();
    _d[ii * length + jj] = d[0][i][j]; 
}

__global__ void floyd_warshall(int *d, int blockSize, int length, 
                            int XIndex, int YIndex, int kk) {
    int ii = blockSize * XIndex + blockIdx.x * blockDim.x + threadIdx.x;
    int jj = blockSize * YIndex + blockIdx.y * blockDim.y + threadIdx.y; 

    int dij = d[ii * length + jj];
    int dik = d[ii * length + kk];
    int dkj = d[kk * length + jj]; 
    
    if (dij > dik + dkj)
        d[ii * length + jj] = dik + dkj;   
}


int main(int argc, char **argv) {
    
    FILE *fin = fopen(argv[1], "r");
    FILE *fout = fopen(argv[2], "w");
    int blockSize = atoi(argv[3]);
     
    int N, M;
    int *edge;
    int *cuda_edge;
    int *cuda_length;
    int *nodeNumber;
    int *index;
    
    fscanf(fin, "%d %d", &N, &M);
    int gridSize = N % blockSize ? N / blockSize + 1 : N / blockSize;
    int length = blockSize * gridSize + 1;
    edge = new int[length * length];
    
    fprintf(stderr, "grid = %d, block = %d\n", gridSize, blockSize);

    for (int i = 0; i < length; ++i) {
        for (int j = 0; j < length; ++j)
            edge[i * length + j] = INF;
        edge[i * length + i] = 0;
    }
    while (M--) {
        int a, b, w;
        fscanf(fin, "%d %d %d", &a, &b, &w);
        a = a - 1;
        b = b - 1;
        edge[a * length + b] = w;
    }
    hipSetDevice(0);
           
    hipMalloc((void**)&cuda_edge, sizeof(int) * length * length);
    cudaCheckErrors("malloc cuda_edge");

    hipMalloc((void**)&nodeNumber, sizeof(int));
    cudaCheckErrors("malloc cuda nodeNumber");

    hipMalloc((void**)&cuda_length, sizeof(int));
    cudaCheckErrors("malloc cuda_length");
    
    hipMalloc((void**)&index, sizeof(int));

    hipMemcpy(cuda_edge, edge, sizeof(int) * length * length, H2D);
    cudaCheckErrors("copy cuda_edge");
    
    hipMemcpy(nodeNumber, &N, sizeof(int), H2D);
    cudaCheckErrors("copy nodeNumber");
    
    hipMemcpy(cuda_length, &length, sizeof(int), H2D);
    cudaCheckErrors("copy cuda_length");

    // Now only hangle N = 3200 testcase
    size_t sharedSize = 40 * 40 ;
    int blockNum = (N + blockSize - 1) / blockSize;
    int blockDimension = 8; 
    gridSize = blockSize / blockDimension;
    
    dim3 blocks(gridSize, gridSize);
    dim3 threads(blockDimension, blockDimension);

    wcout << "blocknum = " << blockNum << endl; 
    for (int k = 0; k < blockNum; ++k) {
        wcout << "k = " << k << endl;
        // phase one
        for (int cur = 0; cur < blockSize; ++cur) {
            floyd_warshall<<<blocks, threads>>>
                (cuda_edge, blockSize, length, k, k, k * blockSize + cur);
        }
        // phase two
        for (int i = 0; i < blockNum; ++i) {
            if (i != k) {
                for (int cur = 0; cur < blockSize; ++cur)
                    floyd_warshall<<<blocks, threads>>>
                            (cuda_edge, blockSize, length, i, k, k * blockSize + cur);
            }
        }
        for (int j = 0; j < blockNum; ++j) {
            if (j != k) {
                for (int cur = 0; cur < blockSize; ++cur)
                    floyd_warshall<<<blocks, threads>>>
                            (cuda_edge, blockSize, length, k, j, k * blockSize + cur);
            }
        }
        
        //phase three
        for (int i = 0; i < blockNum; ++i)
            for (int j = 0; j < blockNum; ++j)
                if (i != k && j != k)
                    for (int cur = 0; cur < blockSize; ++cur)
                        floyd_warshall<<<blocks, threads, 3 * sharedSize>>>
                                (cuda_edge, blockSize, length, i, j, k * blockSize + cur);
    }
    hipMemcpy(edge, cuda_edge, sizeof(int) * length * length, D2H);
    
    for (int i = 0; i < N; ++i) {
        for (int j = 0; j < N - 1; ++j) {
            if (edge[i * length + j] == INF)
                fprintf(fout, "INF ");
            else
                fprintf(fout, "%d ", edge[i * length + j]);
        }
        if (edge[i * length + N - 1] == INF)
            fprintf(fout, "INF\n");
        else
            fprintf(fout, "%d\n", edge[i * length + N - 1]);
    }
    return 0;
}
