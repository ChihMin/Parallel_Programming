#include "hip/hip_runtime.h"
#include <mpi.h>
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <iostream>
#include <string.h>
#include <unistd.h>

#define INF 1e9
#define H2D hipMemcpyHostToDevice
#define D2H hipMemcpyDeviceToHost
#define D2D hipMemcpyDeviceToDevice
#define COMM MPI_COMM_WORLD

#define send(buffer, count, dest) \
    MPI_Send(buffer, count, MPI_CHAR, dest, 0, COMM)

#define recv(buffer, count, src, status)\
    MPI_Recv(buffer, count, MPI_CHAR, src, MPI_ANY_TAG, COMM, &status)

#define MASK_N 2
#define MASK_X 5
#define MASK_Y 5
#define SCALE  8
#define ROOT 0

#define cudaCheckErrors(msg) \
    do { \
        hipError_t __err = hipGetLastError(); \
        if (__err != hipSuccess) { \
            fprintf(stderr, "Fatal error: %s (%s at %s:%d)\n", \
                msg, hipGetErrorString(__err), \
                __FILE__, __LINE__); \
            fprintf(stderr, "*** FAILED - ABORTING\n"); \
            exit(1); \
        } \
    } while (0)

using namespace std;

void DisplayHeader()
{
    const int kb = 1024;
    const int mb = kb * kb;
    wcout << "NBody.GPU" << endl << "=========" << endl << endl;

    //wcout << "CUDA version:   v" << CUDART_VERSION << endl;    
    //wcout << "Thrust version: v" << THRUST_MAJOR_VERSION << "." << THRUST_MINOR_VERSION << endl << endl; 

    int devCount;
    hipGetDeviceCount(&devCount);
    wcout << "CUDA Devices: " << endl << endl;

    for(int i = 0; i < devCount; ++i)
    {
        hipDeviceProp_t props;
        hipGetDeviceProperties(&props, i);
        wcout << i << ": " << props.name << ": " << props.major << "." << props.minor << endl;
        wcout << "  Global memory:   " << props.totalGlobalMem / mb << "mb" << endl;
        wcout << "  Shared memory:   " << props.sharedMemPerBlock / kb << "kb" << endl;
        wcout << "  Constant memory: " << props.totalConstMem / kb << "kb" << endl;
        wcout << "  Block registers: " << props.regsPerBlock << endl << endl;

        wcout << "  Warp size:         " << props.warpSize << endl;
        wcout << "  Threads per block: " << props.maxThreadsPerBlock << endl;
        wcout << "  Max block dimensions: [ " << props.maxThreadsDim[0] << ", " << props.maxThreadsDim[1]  << ", " << props.maxThreadsDim[2] << " ]" << endl;
        wcout << "  Max grid dimensions:  [ " << props.maxGridSize[0] << ", " << props.maxGridSize[1]  << ", " << props.maxGridSize[2] << " ]" << endl;
        wcout << endl;
    }
}

__global__ void floyd_warshall(int *d, int blockSize, int length, 
                            int XIndex, int YIndex, int kk) {
    int ii = blockSize * XIndex + blockIdx.x * blockDim.x + threadIdx.x;
    int jj = blockSize * YIndex + blockIdx.y * blockDim.y + threadIdx.y; 

    int dij = d[ii * length + jj];
    int dik = d[ii * length + kk];
    int dkj = d[kk * length + jj];
    
    if (dij > dik + dkj)
       d[ii * length + jj] = dik + dkj;   
}


int main(int argc, char **argv) {
    
    int rank, size;
    MPI_Init(&argc, &argv);
    MPI_Comm_rank(MPI_COMM_WORLD, &rank);
    MPI_Comm_size(MPI_COMM_WORLD, &size);
    fprintf(stderr, "rank = %d, size = %d\n", rank, size);
     
    MPI_Status status;

    int blockSize = atoi(argv[3]);
     
    int N, M;
    int *edge;
    int *gpu[2];
    int gridSize;
    int length;
    const int deviceNum = 2;
    
    if (rank == ROOT) {
        FILE *fin = fopen(argv[1], "r");
        
        fscanf(fin, "%d %d", &N, &M);
        gridSize = N % blockSize ? N / blockSize + 1 : N / blockSize;
        length = blockSize * gridSize + 1;
        hipHostMalloc((void**)&edge, sizeof(int) * length * length);
        for (int i = 0; i < length; ++i) {
            for (int j = 0; j < length; ++j)
                edge[i * length + j] = INF;
            edge[i * length + i] = 0;
        }
        while (M--) {
            int a, b, w;
            fscanf(fin, "%d %d %d", &a, &b, &w);
            a = a - 1;
            b = b - 1;
            edge[a * length + b] = w;
        }
        send(&N, sizeof(int), 1);
        send(&gridSize, sizeof(int), 1);
        send(&length, sizeof(int), 1);
        send(edge, sizeof(int) * length * length, 1);

        fclose(fin);
    } else {
        recv(&N, sizeof(int), ROOT, status);
        recv(&gridSize, sizeof(int), ROOT, status);
        recv(&length, sizeof(int), ROOT, status);
        hipHostMalloc((void**)&edge, sizeof(int) * length * length);
        recv(edge, sizeof(int) * length * length, ROOT, status);
        fprintf(stderr, "RANK %d, length = %d, gridSize = %d\n", rank, length, gridSize);      
    }
    
    hipSetDevice(rank);
    hipMalloc((void**)&gpu[rank], sizeof(int) * length * length);
    hipMemcpy(gpu[rank], edge, sizeof(int) * length * length, H2D);
    cudaCheckErrors("memcpy error");   
    
    size_t sharedSize = 8 * 8;
    int blockNum = (N + blockSize - 1) / blockSize;
    int blockDimension = 8; 
    gridSize = blockSize / blockDimension;
    int gridFactor = 1024 / blockSize ;
    gridFactor *= gridFactor;

    dim3 blocks(gridSize, gridSize);
    dim3 threads(blockDimension, blockDimension);
    
    dim3 blockCol(gridSize * gridFactor, gridSize);
    dim3 blockRow(gridSize, gridSize * gridFactor);
    int remainBegin = (blockNum / gridFactor) * gridFactor ;  
    int remain = blockNum - remainBegin;
    dim3 blockColRemain(gridSize * remain, gridSize);
    dim3 blockRowRemain(gridSize, gridSize * remain);
    
    fprintf(stderr, "rank %d, gpu = %p \n", rank, gpu[rank]);   
/*
    if (rank == 0) { 
        send(&gpu[0], sizeof(int*), 1);
    }
    else {
        recv(&gpu[0], sizeof(int*), 0, status);
    }
    if (rank != 0) {
        fprintf(stderr, "gpu[0] = %p, gpu[1] = %p", gpu[0], gpu[1]); 
        floyd_warshall<<<blocks, threads>>>(gpu[0], blockSize, length, 0, 0, 0); 
        cudaCheckErrors("launch kernel");
     }
*/
/*
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    
    hipEventRecord(start);
    int n = 5120 / 256;
    for (int i = 0; i < n; ++i) {
        if (rank == 0) {
            hipMemcpy(edge, gpu[0], sizeof(int) * length * length, D2H);
            send(edge, sizeof(int) * length * length, 1);
            recv(edge, sizeof(int) * length * length, 1, status);
            hipMemcpy(gpu[0], edge, sizeof(int) * length * length, H2D);
        } else {
            recv(edge, sizeof(int) * length * length, 0, status);
            hipMemcpy(gpu[1], edge, sizeof(int) * length * length, H2D);
            hipMemcpy(edge, gpu[1], sizeof(int) * length * length, D2H);
            send(edge, sizeof(int) * length * length, 0);
        }
        cudaCheckErrors("memcpy D2D");
    }
    hipEventRecord(stop);
    hipEventSynchronize(stop);

    float time;
    hipEventElapsedTime(&time, start, stop);
    fprintf(stderr, "time = %f\n", time);
*/
    for (int k = 0; k < blockNum; ++k) {
        //wcout << "rank = " << rank <<  "k = " << k  << ", blocknum = " << blockNum << endl;
        // phase one
        {
            for (int cur = 0; cur < blockSize; ++cur) {
                floyd_warshall<<<blocks, threads>>>
                    (gpu[rank], blockSize, length, k, k, k * blockSize + cur);
                cudaCheckErrors("phase one");
            }
        }
        // phase two
        {
            // Column 
            for (int i = 0; i < blockNum - remain; i = i + gridFactor) {
                for (int cur = 0; cur < blockSize; cur++) {
                    floyd_warshall<<<blockCol, threads>>>
                            (gpu[rank], blockSize, length, i, k, k * blockSize + cur);
                    cudaCheckErrors("phase two column main");
                }       
            }
        
            if (remainBegin < blockNum)
                for (int cur = 0; cur < blockSize; cur++) {
                    floyd_warshall<<<blockColRemain, threads>>>
                            (gpu[rank], blockSize, length, remainBegin, k, k * blockSize + cur);
                    cudaCheckErrors("phase two column remain");
                }
            // Row 
            for (int j = 0; j < blockNum - remain; j = j + gridFactor) {
                for (int cur = 0; cur < blockSize; ++cur) {
                    floyd_warshall<<<blockRow, threads>>>
                            (gpu[rank], blockSize, length, k, j, k * blockSize + cur);
                    cudaCheckErrors("phase two row main");
                }
            }
            if (remainBegin < blockNum)
                for (int cur = 0; cur < blockSize; cur++) {
                    floyd_warshall<<<blockRowRemain, threads>>>
                            (gpu[rank], blockSize, length, k, remainBegin, k * blockSize + cur);
                    cudaCheckErrors("phase two row remain");
                }
        }   
        
        //phase three
        {
            int thread = rank;
            int begin, end;
            hipSetDevice(thread);
            
            if (thread == 0) {
                begin = 0;
                end = blockNum / 2;
            } else {
                begin = blockNum / 2;
                end = blockNum;
            }
            for (int i = begin; i < end; i++) {
                for (int j = 0; j < blockNum - remain; j = j + gridFactor) {
                    for (int cur = 0; cur < blockSize; ++cur) {
                        floyd_warshall<<<blockRow, threads>>>
                                (gpu[thread], blockSize, length, i, j, k * blockSize + cur);
                        cudaCheckErrors("phase three row main");
                    }
                }
                if (remainBegin < blockNum)
                    for (int cur = 0; cur < blockSize; cur++) {
                        floyd_warshall<<<blockRowRemain, threads>>>
                                (gpu[thread], blockSize, length, i, remainBegin, k * blockSize + cur);
                        cudaCheckErrors("phase three row remain");
                    }
            }    
        }
        int offset = (blockNum / 2) * blockSize * length ; 
        int copySize = length * length - offset;
        if (rank == 0) {
            hipMemcpy(edge, gpu[0], sizeof(int) * offset, D2H);
            send(edge, sizeof(int) * offset, 1);
            recv(edge + offset, sizeof(int) * copySize, 1, status);
            hipMemcpy(gpu[0] + offset, edge + offset, sizeof(int) * copySize, H2D);
            cudaCheckErrors("rank ROOT memcpy D2D");
        } else {
            recv(edge, sizeof(int) * offset, ROOT, status);
            hipMemcpy(gpu[1], edge, sizeof(int) * offset, H2D);
            hipMemcpy(edge + offset, gpu[1] + offset, sizeof(int) * copySize, D2H);
            cudaCheckErrors("rank 1 memcpy D2D");
            send(edge + offset, sizeof(int) * copySize, ROOT);
        }
    }
    if (rank == 0) {
        FILE *fout = fopen(argv[2], "w");
        for (int i = 0; i < N; ++i) {
            for (int j = 0; j < N - 1; ++j) {
                if (edge[i * length + j] == INF)
                    fprintf(fout, "INF ");
                else
                    fprintf(fout, "%d ", edge[i * length + j]);
            }
            if (edge[i * length + N - 1] == INF)
                fprintf(fout, "INF\n");
            else
                fprintf(fout, "%d\n", edge[i * length + N - 1]);
       }
       fclose(fout);
    }
        
    MPI_Barrier(MPI_COMM_WORLD);
    MPI_Finalize();   
    return 0;
}
